#include "hip/hip_runtime.h"
/* REDUCE KERNEL.cu
 *   by Anonymous
 *
 * Created:
 *   5/24/2020, 9:25:20 PM
 * Last edited:
 *   5/26/2020, 12:00:40 AM
 * Auto updated?
 *   Yes
 *
 * Description:
 *   This file tests and implements te reduction kernel as seen by the CUDA
 *   slides: https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
**/

#include <stdio.h>
#include <stddef.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>


__global__ void reduceKernel(unsigned long* result, unsigned long* to_reduce, size_t N) {
    // Make sure we are allowed to do work
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        // Step 1: fetch global memory to a local shared cache for this block alone

        // First, we allocate shared dynamic memory in the block memory
        extern __shared__ unsigned long cache[];

        // Then, we load our part of the job in it
        int tid = threadIdx.x;
        cache[tid] = to_reduce[i];

        // Make sure also threads outside of our warp did their thing
        __syncthreads();

        // Step 2: do the reduction

        // In our local, superfast cache, let's reduce it
        size_t cache_width;
        if (threadIdx.x != blockDim.x - 1) {
            // Normal width
            cache_width = blockDim.x;
        } else {
            // Reduced width, as there may be caches missing
            cache_width = N % blockDim.x;
            if (cache_width == 0) {
                // This would be impossible, as we won't have a block which has to do 0 elements
                cache_width = blockDim.x;
            }
        }
        
        for (int s = cache_width/2; s > 0; s>>=1) {
            if (tid < s) {
                cache[tid] += cache[tid + s];
            }
            __syncthreads();
        }

        // Step 3: Write the cache memory back
        if (tid == 0) { result[blockIdx.x] = cache[0]; }
    }
}


int main() {
    struct timeval start, stop;

    // Get us a random seed
    srand(time(NULL));

    // Create a list of elements to reduce (make it large, for fun)
    size_t N = 50000;
    int max = 50;
    unsigned long to_reduce[N];
    for (size_t i = 0; i < N; i++) {
        to_reduce[i] = rand() % max;
    }

    // Acquire a correct value, and benchmark the sequential version while we're at it
    gettimeofday(&start, NULL);
    
    unsigned long correct = 0;
    for (size_t i = 0; i < N; i++) {
        correct += to_reduce[i];
    }

    gettimeofday(&stop, NULL);

    unsigned long time_taken = ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;
    printf("Sequential time taken: %lu ms\n", time_taken);
    printf("Sequential result: %lu\n", correct);




    
    // Now, enter CUDA!

    gettimeofday(&start, NULL);

    // Allocate space for N + copy the data
    unsigned long* to_reduce_gpu;
    hipMalloc(&to_reduce_gpu, sizeof(unsigned long) * N);
    hipMemcpy(to_reduce_gpu, to_reduce, sizeof(unsigned long) * N, hipMemcpyHostToDevice);

    // Allocate space for the result
    unsigned long* result_gpu;
    hipMalloc(&result_gpu, sizeof(unsigned long) * N);

    // Next, invoke the kernel as many times as needed. Let's say that we do the rest manually from 32 and down
    int threadsPerBlock = 32;
    int to_go = N;
    while (to_go > 32) {
        int blocksPerGrid = to_go / threadsPerBlock + (to_go % threadsPerBlock == 0 ? 0 : 1);
        reduceKernel<<<blocksPerGrid, threadsPerBlock, sizeof(unsigned long) * threadsPerBlock>>>(result_gpu, to_reduce_gpu, to_go);

        to_go = blocksPerGrid;

        printf("Next round:\n");
        printf("to_go=%d\n", to_go);
        hipMemcpy(to_reduce, to_reduce_gpu, sizeof(unsigned long) * N + to_pad, hipMemcpyDeviceToHost);
        printf("Elements of list: [");
        for (size_t i = 0; i < N + to_pad; i++) {
            if (i > 0) { printf(", "); }
            printf("%lu", to_reduce[i]);
        }
        printf("] (%lu long)\n", N + to_pad);

        // Swap dem pointers
        unsigned long* temp = result_gpu;
        result_gpu = to_reduce_gpu;
        to_reduce_gpu = temp;
    }

    // Copy the memory back
    hipMemcpy(to_reduce, result_gpu, sizeof(unsigned long) * to_go, hipMemcpyDeviceToHost);

    // Manually combine all intermediate results
    unsigned long result = 0;
    for (size_t i = 0; i < to_go; i++) {
        result += to_reduce[i];
    }

    hipFree(to_reduce_gpu);
    hipFree(result_gpu);

    gettimeofday(&stop, NULL);

    time_taken = ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;
    printf("CUDA time taken: %lu ms\n", time_taken);
    printf("CUDA result: %lu\n", result);

    return 0;
}

